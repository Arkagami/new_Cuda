#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#include <stdio.h>
#include <malloc.h>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_DIM 16

__global__ void transposition(int* matrix, int* matrixOut, int length, int width)
{
	__shared__ int tempMatrix[BLOCK_DIM][BLOCK_DIM];//����������� ������

	int temp;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if ((i < length) && (j < width))
	{
		temp = j * length + i;
		tempMatrix[threadIdx.y][threadIdx.x] = matrix[temp];
	}

	__syncthreads();

	i = blockIdx.y * blockDim.y + threadIdx.x;//������ �����, ����������� �����, ������ ������
	j = blockIdx.x * blockDim.x + threadIdx.y;

	if ((i < width) && (j < length))
	{
		temp = j * width + i;
		matrixOut[temp] = tempMatrix[threadIdx.x][threadIdx.y];
	}
}

int main()
{
	int length, width, *matrixInput, *matrixOut, size, *devMatrixInput, *devOutputMatrix;
	while (1) {
		printf("Input width:");
		scanf("%d", &width);
		printf("Input length:");
		scanf("%d", &length);
		size = length * width * sizeof(int);
		matrixInput = (int *)malloc(size);
		matrixOut = (int *)malloc(size);
		for (int i = 0; i < (length * width); i++)
		{
			matrixInput[i] = i + 1;
		}
		printf("Input matrix:\n--------------------------------------------------------------------------------------\n");
		for (int i = 0; i < width; i++)
		{
			printf("|");
			for (int j = 0; j < length; j++)
			{
				printf("%-4d|", matrixInput[i*length + j]);
			}
			printf("\n--------------------------------------------------------------------------------------\n");
		}
		printf("\n\n\n");

		hipMalloc((void**)&devMatrixInput, size);
		hipMalloc((void**)&devOutputMatrix, size);

		hipMemcpy(devMatrixInput, matrixInput, size, hipMemcpyHostToDevice);

		dim3 dimBlock(BLOCK_DIM, BLOCK_DIM, 1);
		dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (length + dimBlock.y - 1) / dimBlock.y, 1);

		transposition<<<dimGrid, dimBlock>>>(devMatrixInput, devOutputMatrix, length, width);

		hipDeviceSynchronize();
		hipMemcpy(matrixOut, devOutputMatrix, size, hipMemcpyDeviceToHost);

		hipFree(devMatrixInput);
		hipFree(devOutputMatrix);

		printf("Output matrix:\n--------------------------------------------------------------------------------------\n");
		for (int i = 0; i < length; i++)
		{
			printf("|");
			for (int j = 0; j < width; j++)
			{
				printf("%-3d|", matrixOut[i*width + j]);
			}
			printf("\n--------------------------------------------------------------------------------------\n");
		}
		printf("\n\n\n");

		free(matrixInput);
		free(matrixOut);
	}
	return 0;
}